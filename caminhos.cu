// ----------------------------------------------------------------------------
// Dado um grafo direcionado com pesos nas arestas, encontrar distância do caminho mais curto entre cada par de vértices.
// Supor:
//		Arestas possuem pesos > 0
//		Grafo não possui loops (aresta de um vértice para ele mesmo)
//		Número de vértices do grafo é potência de 2
//
// Para compilar: gcc caminhos_seq.c -o caminhos_seq
// Para executar: ./caminhos_seq nVértices arquivoEntrada arquivoSaída


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>

// ----------------------------------------------------------------------------
void leArqIn(char *nomeArqIn, int nElem, unsigned int *mat)
{
	unsigned int p;

	FILE *arqIn = fopen(nomeArqIn, "rt");	// Arquivo texto de entrada

	if (arqIn == NULL)
	{
		printf("\nArquivo texto de entrada não encontrado\n");
		exit(1);
	}

	// Lê matriz nElem x nElem de adjacências do arquivo de entrada
	// Inicializa matriz nElem x nElem de distâncias com distâncias mínimas de caminhos com até 1 aresta
	for (int i = 0; i < nElem; i++)
		for (int j = 0; j < nElem; j++)
		{
			fscanf(arqIn, "%u", &p); // peso da aresta (i,j)
			// Peso 0 indica ausência de aresta de i para j
			if (p == 0 && i != j)
				mat[i*nElem + j] = INT_MAX; // mat[i][j] = infinito
			else
				mat[i*nElem + j] = p; // mat[i][j] = adj[i][j] = peso da aresta (i,j)
		}

	fclose(arqIn);
}

// ----------------------------------------------------------------------------
void calculaDist(int nElem, unsigned int *matIn, unsigned int *matOut)
{
	unsigned int dij, dikj;

	// Para cada par de vértices i e j
	for (int i = 0; i < nElem; i++)
		for (int j = 0; j < nElem; j++)
		{
			// Calcula distância mínima de i para j, testando,
			// para cada vértice k, distância de i para k + distância de k para j
			dij = INT_MAX;
			for (int k = 0; k < nElem; k++)
			{
				dikj = matIn[i*nElem + k] + matIn[k*nElem + j]; // matIn[i][k] + matIn[k][j]
				if (dikj < dij)
					dij = dikj;
			}
			matOut[i*nElem + j] = dij; // matOut[i][j]
		}
}

// ----------------------------------------------------------------------------
void escreveArqOut(char* nomeArqOut, int nElem, unsigned int *mat)
{
	FILE *arqOut;	// Arquivo texto de saída

	arqOut = fopen(nomeArqOut, "wt");

	// Escreve matriz nElem x nElem de distâncias no arquivo de saída
	for (int i = 0; i < nElem; i++)
	{
		for (int j = 0; j < nElem; j++)
			if (mat[i*nElem + j] != INT_MAX)	// mat[i][j]
				fprintf(arqOut, "%d ", mat[i*nElem + j]); // Distância mínima de i para j
			else
				fprintf(arqOut, "-1 ");	// Não há caminho de i para j
		fprintf(arqOut, "\n");
	}

	fclose(arqOut);
}

// ----------------------------------------------------------------------------
// Programa principal
int main(int argc, char** argv)
{
	if(argc != 4)
	{
		printf("O programa foi executado com argumentos incorretos.\n");
		printf("Uso: ./caminhos_seq nVértices arquivoEntrada arquivoSaída\n");
		exit(1);
	}

	int nVert = atoi(argv[1]);	// Obtém número de vértices do grafo

	// Obtém nome dos arquivos de entrada e saída
	char nomeArqIn[100],
		  nomeArqOut[100] ;

	strcpy(nomeArqIn, argv[2]) ;
	strcpy(nomeArqOut, argv[3]) ;

	// Aloca matrizes nVert x nVert de distâncias mais curtas
	int nBytes = nVert * nVert * sizeof(int);
	unsigned int *distIn = (unsigned int *) malloc(nBytes);
	unsigned int *distOut = (unsigned int *) malloc(nBytes);
	unsigned int *auxTroca;
	if (distIn == NULL || distOut == NULL)
	{
		printf("\nErro na alocação das estruturas de dados\n");
		exit(1);
	}

	// Lê matriz de adjacências do arquivo de entrada e inicializa matriz de distâncias
	leArqIn(nomeArqIn, nVert, distIn);

	// Inicia medição do tempo de execução
	struct timeval h_ini, h_fim;
	gettimeofday(&h_ini, 0);

	// Repete log_2(nVert) passos
	for (int alcance = 2; alcance <= nVert; alcance<<=1)
	{
		// Usando distâncias mínimas de caminhos com até (alcance) arestas,
		// calcula distâncias mínimas de caminhos com até (2*alcance) arestas
		calculaDist(nVert, distIn, distOut);

		// Troca ponteiros de matrizes de distâncias distIn e distOut
		auxTroca = distIn;
		distIn   = distOut;
		distOut  = auxTroca;
	}
	
	// Finaliza medição do tempo de execução
	gettimeofday(&h_fim, 0);
	long segundos = h_fim.tv_sec - h_ini.tv_sec;
	long microsegundos = h_fim.tv_usec - h_ini.tv_usec;
	double h_tempo = (segundos * 1e3) + (microsegundos * 1e-3); // Tempo de execução na CPU em ms

	// Escreve matriz de distâncias no arquivo de saída
	escreveArqOut(nomeArqOut, nVert, distIn);

	printf("Tempo CPU = %.2fms\n", h_tempo);

	// Libera matrizes
	free(distIn);
	free(distOut);

	return 0;
}
