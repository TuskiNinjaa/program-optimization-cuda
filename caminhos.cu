// ----------------------------------------------------------------------------
// Nome dos alunos do grupo:
// Vitor Yuske Watanabe - 2020.1905.058-4
// Raissa Rinaldi Yoshioka - 2020.1905.049-5
// ----------------------------------------------------------------------------
// Dado um grafo direcionado com pesos nas arestas, encontrar distância do caminho mais curto entre cada par de vértices.
// Supor:
//		Arestas possuem pesos > 0
//		Grafo não possui loops (aresta de um vértice para ele mesmo)
//		Número de vértices do grafo é potência de 2
//
// Para compilar: nvcc caminhos.cu -o caminhos
// Para executar: ./caminhos nVértices arquivoEntrada arquivoSaída


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>

// ----------------------------------------------------------------------------
// Macro para checagem de erro das chamadas às funções do CUDA
#define checa_cuda(result)                          \
    if (result != hipSuccess)                      \
    {                                               \
        printf("%s\n", hipGetErrorString(result)); \
        exit(1);                                    \
    }

// ----------------------------------------------------------------------------
void leArqIn(char *nomeArqIn, int nElem, unsigned int *mat)
{
    unsigned int p;

    FILE *arqIn = fopen(nomeArqIn, "rt"); // Arquivo texto de entrada

    if (arqIn == NULL)
    {
        printf("\nArquivo texto de entrada não encontrado\n");
        exit(1);
    }

    // Lê matriz nElem x nElem de adjacências do arquivo de entrada
    // Inicializa matriz nElem x nElem de distâncias com distâncias mínimas de caminhos com até 1 aresta
    for (int i = 0; i < nElem; i++)
        for (int j = 0; j < nElem; j++)
        {
            fscanf(arqIn, "%u", &p); // peso da aresta (i,j)
            // Peso 0 indica ausência de aresta de i para j
            if (p == 0 && i != j)
                mat[i * nElem + j] = INT_MAX; // mat[i][j] = infinito
            else
                mat[i * nElem + j] = p; // mat[i][j] = adj[i][j] = peso da aresta (i,j)
        }

    fclose(arqIn);
}

// ----------------------------------------------------------------------------
__global__ void calculaDist(int nElem, unsigned int *matIn, unsigned int *matOut)
{
	// linha i = coordenada y do id GLOBAL da thread
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	// coluna j = coordenada x do id GLOBAL da thread
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	// Se thread corresponde a uma célula da matriz C
	if ((i < nElem) && (j < nElem))
	{
        unsigned int dij = INT_MAX, dikj;
        for (int k = 0; k < nElem; k++)
        {
            dikj = matIn[i * nElem + k] + matIn[k * nElem + j]; // matIn[i][k] + matIn[k][j]
            if (dikj < dij)
                dij = dikj;
        }
        matOut[i * nElem + j] = dij; // matOut[i][j]
	}
}

// ----------------------------------------------------------------------------
void escreveArqOut(char *nomeArqOut, int nElem, unsigned int *mat)
{
    FILE *arqOut; // Arquivo texto de saída

    arqOut = fopen(nomeArqOut, "wt");

    // Escreve matriz nElem x nElem de distâncias no arquivo de saída
    for (int i = 0; i < nElem; i++)
    {
        for (int j = 0; j < nElem; j++)
            if (mat[i * nElem + j] != INT_MAX)              // mat[i][j]
                fprintf(arqOut, "%d ", mat[i * nElem + j]); // Distância mínima de i para j
            else
                fprintf(arqOut, "-1 "); // Não há caminho de i para j
        fprintf(arqOut, "\n");
    }

    fclose(arqOut);
}

// ----------------------------------------------------------------------------
// Programa principal
int main(int argc, char **argv)
{
    if (argc != 4)
    {
        printf("O programa foi executado com argumentos incorretos.\n");
        printf("Uso: ./caminhos_seq nVértices arquivoEntrada arquivoSaída\n");
        exit(1);
    }

    int nVert = atoi(argv[1]); // Obtém número de vértices do grafo

    // Obtém nome dos arquivos de entrada e saída
    char nomeArqIn[100],
        nomeArqOut[100];

    strcpy(nomeArqIn, argv[2]);
    strcpy(nomeArqOut, argv[3]);

    // Cria variáveis
    unsigned int nBytes = nVert * nVert * sizeof(int), // Tamanho dos vetores em bytes
        *distInHost, *distInDevice, *distOutDevice;

    // Aloca o vetor no host
    distInHost = (unsigned int *)malloc(nBytes);
    if (distInHost == NULL)
    {
        printf("\nErro na alocação das estruturas de dados\n");
        exit(1);
    }

    // Lê matriz de adjacências do arquivo de entrada e inicializa matriz de distâncias
    leArqIn(nomeArqIn, nVert, distInHost);

    // Aloca vetores na memória global da GPU
    checa_cuda(hipMalloc((void **)&distInDevice, nBytes));
    checa_cuda(hipMalloc((void **)&distOutDevice, nBytes));

	// Determina nBlocos e nThreadsBloco
	// nBlocos.x = teto(m / nThreadsBloco.x)
	// nBlocos.y = teto(n / nThreadsBloco.y)
	dim3 nThreadsBloco(32,32);
	dim3 nBlocos((nVert + (nThreadsBloco.x - 1)) / nThreadsBloco.x, (nVert + (nThreadsBloco.y - 1)) / nThreadsBloco.y);

    // Inicia medição de tempo de execução na GPU
    hipEvent_t d_ini, d_fim;
    hipEventCreate(&d_ini);
    hipEventCreate(&d_fim);
    hipEventRecord(d_ini, 0);

    // Copia dados de entrada do host para memória global da GPU
    checa_cuda(hipMemcpy(distInDevice, distInHost, nBytes, hipMemcpyHostToDevice));

    // Repete log_2(nVert) passos
    for (int alcance = 2; alcance <= nVert; alcance <<= 1)
    {
        // Usando distâncias mínimas de caminhos com até (alcance) arestas,
        // calcula distâncias mínimas de caminhos com até (2*alcance) arestas
        calculaDist<<<nBlocos, nThreadsBloco>>>(nVert, distInDevice, distOutDevice);

        // Host aguarda a execução da GPU
        hipDeviceSynchronize();

        // Atribui os valores de distOutDevice em distInDevice
        checa_cuda(hipMemcpy(distInDevice, distOutDevice, nBytes, hipMemcpyDeviceToDevice));
    }

    // Copia dados de entrada da GPU para o host
    checa_cuda(hipMemcpy(distInHost, distInDevice, nBytes, hipMemcpyDeviceToHost));

    // Finaliza medição do tempo de execução
    hipEventRecord(d_fim, 0);
    hipEventSynchronize(d_fim);
    float d_tempo; // Tempo de execução na GPU em milissegundos
    hipEventElapsedTime(&d_tempo, d_ini, d_fim);
    hipEventDestroy(d_ini);
    hipEventDestroy(d_fim);
    printf("Tempo GPU = %.2fms\n", d_tempo);

    // Escreve matriz de distâncias no arquivo de saída
    escreveArqOut(nomeArqOut, nVert, distInHost);

    // Libera vetores na memória global da GPU
    checa_cuda(hipFree(distInDevice));
    checa_cuda(hipFree(distOutDevice));

    // Libera vetor no host
    free(distInHost);

    return 0;
}
